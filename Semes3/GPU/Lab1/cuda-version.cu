#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define THREADSPBLK 1024
#define THREADSPSM 2048
#define TILE_WIDTH 16
#define TOTAL_ITERATIONS 50

int main_n;

__global__ void iterate(float* originalMatrixD, float* solutionD, int originalMatrixWidth, 
        int startingIndex) {
    __shared__ float originalMatrixDS [TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int currentMatrixIndex = blockId * (blockDim.x * blockDim.y) + 
        (threadIdx.y * blockDim.x) + threadIdx.x;

    currentMatrixIndex += startingIndex;

    originalMatrixDS[ty][tx] = originalMatrixD[currentMatrixIndex];

    // Sync up w/ shared data set up
    __syncthreads();

    float replaceAmount;
    bool onEdge = false;
    int XEdgeCheckMod = currentMatrixIndex % originalMatrixWidth;

    // X = 0 edge
    if ( XEdgeCheckMod == 0) {
        //onEdge = true;
        replaceAmount = 11.0;

    }

    // X = N - 1
    else if ( XEdgeCheckMod == (originalMatrixWidth - 1)) {
        //onEdge = true;
        replaceAmount = 22.0;
    }

    // Y = 0
    else if (currentMatrixIndex < originalMatrixWidth) {
        //onEdge = true;
        replaceAmount = 33.0;
    }

    // Y = N - 1
    else if (currentMatrixIndex >= (originalMatrixWidth * originalMatrixWidth 
                - originalMatrixWidth)) {
        //onEdge = true;
        replaceAmount = 44.0;
    }

    else {
        replaceAmount = 55.0;
    }

    /*
    if (onEdge) {
        //replaceAmount = originalMatrixDS[ty][tx];
        //replaceAmount = originalMatrixD[currentMatrixIndex];
        replaceAmount = 55.05;
    }

    else {
        //replaceAmount = (originalMatrixDS[ty+1][tx] + originalMatrixDS[ty-1][tx] +
            // originalMatrixDS[ty][tx+1] + originalMatrixDS[ty][tx-1])/4.0;
        replaceAmount = 88.0;
    }
    */

    solutionD[currentMatrixIndex] = replaceAmount;
    //solutionD[currentMatrixIndex] = originalMatrixD[currentMatrixIndex];
}

void setUp(float *a, int size) {
    for (int y = 0; y < size; y++) {
        for (int x = 0; x < size; x++) {
            if (y == 0) {
                a[y * size + x] = 80;
            }

            else if (y == size - 1) {
                a[y * size + x] = 80;
            }

            else if (x == size - 1) {
                a[y * size + x] = 80;
            }

            else if (x == 0) {
                if ((y >= 10) && (y <= 30)) {
                    a[y * size + x] = 150;
                }
                else {
                    a[y * size + x] = 80;
                }
            }
            else {
                a[y * size + x] = 0;
            }
        }
    }
}

int main ( int argc, char *argv[] )
{
    if (argc == 2) {
        main_n = atoi(argv[1]);
        main_n +=1;
    }
    else {
        main_n += 101;
    }
    
    int elements = main_n * main_n;
    float originalMatrix[elements];
    float solution[elements];

    setUp(originalMatrix, main_n);
   
    // GPU TIME BABY!
    dim3 dimGrid(4, 2);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    float* solutionD;
    float* originalMatrixD;

    int memorySize = main_n * main_n * sizeof(float);
    
    hipMalloc((void**) &originalMatrixD, memorySize);
    hipMalloc((void**) &solutionD, memorySize);

    /*
    for (int i = 0; i < TOTAL_ITERATIONS; i++) {
        cudaMemcpy(originalMatrixD, originalMatrix, memorySize, cudaMemcpyHostToDevice);
     
        // INVOKE
        iterate<<<dimGrid, dimBlock>>>(originalMatrixD, solutionD, main_n);

        // Finish
        cudaMemcpy(solution, solutionD, memorySize, cudaMemcpyDeviceToHost);

        printf("After iteration solution value @ 0 20: %f\n", solution[20 * main_n + 0]);
        printf("After iteration solution value @ 30 30: %f\n", solution[30 * main_n + 30]);
        // copy to solution to OriginalMatrixD for iteration
        memcpy(originalMatrix, solution, memorySize);
    }
    */

    hipMemcpy(originalMatrixD, originalMatrix, memorySize, hipMemcpyHostToDevice);
     
    // INVOKE
    for (int i = 0; i < elements/THREADSPSM + 1; i++) {
        int startingIndex = i * THREADSPSM;
        iterate<<<dimGrid, dimBlock>>>(originalMatrixD, solutionD, main_n, startingIndex);
    }

    // Finish
    hipMemcpy(solution, solutionD, memorySize, hipMemcpyDeviceToHost);

    int counter = 0;
    for (int i = 0; i < main_n * main_n; i++) {
        if (i % main_n == 0) {
            printf("\n\n\n\n\n%d\n%f", counter++, solution[i]);
        }
        else {
            printf(" %f", solution[i]);
        }
    }

    // copy to solution to OriginalMatrixD for iteration
    //memcpy(originalMatrix, solution, memorySize);

    hipFree(originalMatrixD);
    hipFree(solutionD);

    int Y = 10;
    int X = 10;

    float printSolutionAtIndex = solution[Y * main_n + X];
    printf("After iterations, at [10][10] we have: %f\n", printSolutionAtIndex);
    //free(solution);
}



