#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define THREADSPBLK 1024
#define THREADSPSM 2048
#define TILE_WIDTH 32
#define TOTAL_ITERATIONS 50

int main_n;

__global__ void iterate(float* originalMatrixD, float* solutionD, int originalMatrixWidth, 
        int startingIndex) {
    // __shared__ float originalMatrixDS [TILE_WIDTH][TILE_WIDTH];
    __shared__ float originalMatrixDS [TILE_WIDTH * TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int currentMatrixIndex = blockId * (blockDim.x * blockDim.y) + 
        (threadIdx.y * blockDim.x) + threadIdx.x;

    currentMatrixIndex += startingIndex;

    originalMatrixDS[ty * TILE_WIDTH + tx] = originalMatrixD[currentMatrixIndex];

    // Sync up w/ shared data set up
    __syncthreads();

    float replaceAmount;
    bool onEdge = false;
    int XEdgeCheckMod = currentMatrixIndex % originalMatrixWidth;

    // X = 0 edge
    if ( XEdgeCheckMod == 0) {
        onEdge = true;
    }

    // X = N - 1
    else if ( XEdgeCheckMod == (originalMatrixWidth - 1)) {
        onEdge = true;
    }

    // Y = 0
    else if (currentMatrixIndex < originalMatrixWidth) {
        onEdge = true;
    }

    // Y = N - 1
    else if (currentMatrixIndex >= (originalMatrixWidth * originalMatrixWidth 
                - originalMatrixWidth)) {
        onEdge = true;
    }

    if (onEdge) {
        replaceAmount = originalMatrixDS[ty * TILE_WIDTH + tx];
    }

    else {
        // Top and Bottom come from Global memory
        float top = originalMatrixD[currentMatrixIndex - originalMatrixWidth];
        float bottom = originalMatrixD[currentMatrixIndex + originalMatrixWidth];
        float left;
        float right;

        // Left and right edge come from Global memory
        if (tx == 0 && ty == 0) {
            left = originalMatrixD[currentMatrixIndex - 1];
        }

        else {
            left = originalMatrixDS[ty * TILE_WIDTH + tx - 1];
        }

        if ((ty == TILE_WIDTH - 1) && (tx == TILE_WIDTH - 1)) {
            right = originalMatrixD[currentMatrixIndex + 1];
        }

        else {
            right = originalMatrixDS[ty * TILE_WIDTH + tx + 1];
        }

        replaceAmount = (left + right + top + bottom) / 4;
    }

    solutionD[currentMatrixIndex] = replaceAmount;
}

void setUp(float *a, int size) {
    for (int y = 0; y < size; y++) {
        for (int x = 0; x < size; x++) {
            if (y == 0) {
                a[y * size + x] = 80;
            }

            else if (y == size - 1) {
                a[y * size + x] = 80;
            }

            else if (x == size - 1) {
                a[y * size + x] = 80;
            }

            else if (x == 0) {
                if ((y >= 10) && (y <= 30)) {
                    a[y * size + x] = 150;
                }
                else {
                    a[y * size + x] = 80;
                }
            }
            else {
                a[y * size + x] = 0;
            }
        }
    }
}


int main ( int argc, char *argv[] )
{
    if (argc == 2) {
        main_n = atoi(argv[1]);
        main_n +=1;
    }
    else {
        main_n += 101;
    }
    
    int elements = main_n * main_n;
    float *originalMatrix = (float *)malloc(elements * sizeof(float));
    float *solution = (float *)malloc(elements * sizeof(float));
    //float originalMatrix[elements];
    //float solution[elements];

    setUp(originalMatrix, main_n);
   
    // GPU TIME BABY!
    dim3 dimGrid(2, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    
    float* solutionD;
    float* originalMatrixD;

    int memorySize = main_n * main_n * sizeof(float);
    
    hipMalloc((void**) &originalMatrixD, memorySize);
    hipMalloc((void**) &solutionD, memorySize);

    //cudaMemcpy(originalMatrixD, originalMatrix, memorySize, cudaMemcpyHostToDevice);
     
    // INVOKE
    // for (int iter = 0; iter < TOTAL_ITERATIONS; iter++) {
    for (int iter = 0; iter < 50; iter++) {
        for (int i = 0; i < elements/THREADSPSM + 1; i++) {
            int startingIndex = i * THREADSPSM;
            hipMemcpy(originalMatrixD, originalMatrix, memorySize, hipMemcpyHostToDevice);
            iterate<<<dimGrid, dimBlock>>>(originalMatrixD, solutionD, main_n, startingIndex);
            hipMemcpy(originalMatrix, solutionD, memorySize, hipMemcpyDeviceToHost);
        }
    }

    // Finish
    hipMemcpy(solution, solutionD, memorySize, hipMemcpyDeviceToHost);

    // PRINTS
    /*
    int counter = 0;
    for (int i = 0; i < main_n * main_n; i++) {
        if (i % main_n == 0) {
            printf("\n\n\n\n\n%d\n%f", counter++, solution[i]);
        }
        else {
            printf(" %f", solution[i]);
        }
    }
    */

    hipFree(originalMatrixD);
    hipFree(solutionD);
    free(originalMatrix);
    free(solution);
}
